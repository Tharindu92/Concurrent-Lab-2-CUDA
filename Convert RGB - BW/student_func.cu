#include "hip/hip_runtime.h"
// Homework 1
// Color to Greyscale Conversion

//A common way to represent color images is known as RGBA - the color
//is specified by how much Red, Green, and Blue is in it.
//The 'A' stands for Alpha and is used for transparency; it will be
//ignored in this homework.

//Each channel Red, Blue, Green, and Alpha is represented by one byte.
//Since we are using one byte for each color there are 256 different
//possible values for each color.  This means we use 4 bytes per pixel.

//Greyscale images are represented by a single intensity value per pixel
//which is one byte in size.

//To convert an image from color to grayscale one simple method is to
//set the intensity to the average of the RGB channels.  But we will
//use a more sophisticated method that takes into account how the eye 
//perceives color and weights the channels unequally.

//The eye responds most strongly to green followed by red and then blue.
//The NTSC (National Television System Committee) recommends the following
//formula for color to greyscale conversion:

//I = .299f * R + .587f * G + .114f * B

//Notice the trailing f's on the numbers which indicate that they are 
//single precision floating point constants and not double precision
//constants.

//You should fill in the kernel as well as set the block and grid sizes
//so that the entire image is processed.

#include "reference_calc.cpp"
#include "utils.h"
#include <stdio.h>

__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
  /*
   * Find the location of the image pixel
   */
  int x = (blockIdx.x * blockDim.x) + threadIdx.x; //row id of the pixel
  int y = (blockIdx.y * blockDim.y) + threadIdx.y; //column id of the pixel
  
  /*
   * Get the rgb value of original image for the location above found
   */
  uchar4 rgb_value = *(rgbaImage + (x * numCols + y));
  
  /*
   * Calculate grey scale pixel value for the same location 
   */
  unsigned char grey_value = rgb_value.x*.299f + rgb_value.y*.587f + rgb_value.z*.114f;
  
  /*
   * Save the calculated value for memory allocated of gray scale image
   */
  *(greyImage + (x * numCols + y)) = grey_value;
  
}

__global__
void lightness_rgba_to_greyscale(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
  /*
   * Find the location of the image pixel
   */
  int x = (blockIdx.x * blockDim.x) + threadIdx.x; //row id of the pixel
  int y = (blockIdx.y * blockDim.y) + threadIdx.y; //column id of the pixel
  
  /*
   * Get the rgb value of original image for the location above found
   */
  uchar4 rgb_value = *(rgbaImage + (x * numCols + y));
  
  /*
   * Calculate grey scale pixel value for the same location 
   */
  unsigned char max_rgba = rgb_value.x;
  unsigned char min_rgba = rgb_value.x;
  
  if(rgb_value.y > max_rgba){
    max_rgba = rgb_value.y;
  }
  
  if(rgb_value.z > max_rgba){
    max_rgba = rgb_value.z;
  }
  
  if(rgb_value.y < min_rgba){
    min_rgba = rgb_value.y;
  }
  
  if(rgb_value.z < min_rgba){
    min_rgba = rgb_value.z;
  }
  
  unsigned char grey_value = (max_rgba + min_rgba)/2;
  
  /*
   * Save the calculated value for memory allocated of gray scale image
   */
  *(greyImage + (x * numCols + y)) = grey_value;
}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{
  /*
   * Divide rows into 16 and columns into 16. 
   */ 
  int choise; 
  const dim3 blockSize(17, 17, 1);  //TODO
  const dim3 gridSize( numRows/16, numCols/16, 1);  //TODO
  rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
  lightness_rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}
